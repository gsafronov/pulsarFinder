#include "hip/hip_runtime.h"
#include "include/DMScan.h"
#include <hip/hip_runtime.h>

//before class definition define CUDA kernel func
//see comments on actions in DMScan::sumFrequencies() function
//treat array as 1dim: in thread i sum_j(511-j+i), fullID - ID on a given freq band
__global__ void sumFreq_kernel(float *sigArray, float *sigSum, int nFreq, int nBins, int nBinsPerPeriod, float DM, float l511, float dL, float tau, float period)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  float bico=0;
  int fullID=i;
  for (int iFreq=0; iFreq<nFreq; iFreq++){
    float dT=4.6*(-l511*l511+(l511+iFreq*dL)*(l511+iFreq*dL))*DM*0.001;
    float dTnearest=dT-period*floor(dT/period);
    float delta=dTnearest/tau;
    float bico1=sigArray[((511-iFreq)*nBins+int(floor(fullID+delta)))%(nBins*nFreq)];
    float bico2=sigArray[((511-iFreq)*nBins+int((floor(fullID+delta)+1)))%(nBins*nFreq)];
    float loFrac=1-((fullID+delta)-floor(fullID+delta));
    float upFrac=1-loFrac;
    bico+=loFrac*bico1+upFrac*bico2;
  }
  sigSum[fullID]=bico;
}

int DMScan::loadDataToGPU()
{
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  // Print the vector length to be used, and compute its size
  size_t size_input=(nBins*nFreq)*sizeof(float);
  size_t size_output=(nBins)*sizeof(float);	
  
  // Allocate the device input signal vector 
  err = hipMalloc((void **)&d_sigArray, size_input);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate the device input signal vector (error code %s)!\n", hipGetErrorString(err));
    
    free(sigArray);
    
    return 0;
    //exit(EXIT_FAILURE);
  }
  
  // Copy the host input signal vector to the device signal vector in device memory
  err = hipMemcpy(d_sigArray, sigArray, size_input, hipMemcpyHostToDevice);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy input signal vector from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  free(sigArray);
  return 0;
}

int DMScan::sumFrequencies_GPU(int iStep)
{
  float DM=DM0+scanStep*iStep;
  
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  size_t size_output=nBins*sizeof(float);

  //host compensated DM
  float* sigSum=(float *)malloc(size_output);
  
  // device compensated DM
  float* d_sigSum;
  err = hipMalloc((void **)&d_sigSum, size_output);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate compensated signal vector on the device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 32;
  int blocksPerGrid = floor(nBins/threadsPerBlock)+1;
  //  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  //       std::cout<<"parameters: "<<nFreq<<"  "<<nBins<<"  "<<DM<<"  "<<l511<<"  "<<dL<<"  "<<tau<<"  "<<period<<"   blocksPerGrid "<<blocksPerGrid<<std::endl;
  sumFreq_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_sigArray, d_sigSum, nFreq, nBins, nBinsPerPeriod, DM, l511, dL, tau, period);
  err = hipGetLastError();
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to launch sumFreq kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  // Copy the device result vector in device memory to the host result vector in host memory.
  //  printf("Copy output data from the CUDA device to the host memory:");
  //     std::cout<<"address: "<<sigSum<<"  "<<&sigSum[0]<<"  "<<d_sigSum<<"  "<<size_output<<"  "<<hipMemcpyDeviceToHost<<std::endl;
  err = hipMemcpy(sigSum, d_sigSum, size_output, hipMemcpyDeviceToHost);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy compensated signal vector from device to host (error code: %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  for (int i=0; i<nBinsGlobal; i++){
    if (sigSum[i]==sigSum[i]&&i<nBins) sumFreq[iStep]->SetBinContent(i+1,sigSum[i]);
    else sumFreq[iStep]->SetBinContent(i+1,512);
  }
  
  //allocate host vector with non-zero elements and image
  //use common number of bins for all runs (nBinsGlobal)
  
  int signal_memSize=sizeof(float)*nBinsGlobal;
  float* h_sigSum_NZ = (float*)malloc(signal_memSize);
  //fill the vector
  for (int i=0; i<nBinsGlobal; i++){
    if (sigSum[i]!=0&&sigSum[i]==sigSum[i]&&i<nBins) h_sigSum_NZ[i]=sigSum[i];
    else h_sigSum_NZ[i]=512;
  }
  
  //allocate device vector with non-zero elements:
  int image_memSize = sizeof(float2) * (floor(nBinsGlobal/2)+1);
  float* d_sigSum_NZ;
  err=hipMalloc((void **)&d_sigSum_NZ, image_memSize);
  if (err!=hipSuccess){			
    std::cout<<"Failed to allocate device signal NZ vector"<<std::endl; 
    return 1;
  }
  
  //copy host to device
  err = hipMemcpy(d_sigSum_NZ, h_sigSum_NZ, image_memSize, hipMemcpyHostToDevice);

  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy signal NZ vector from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  /////////
  
  //allocate host and device images
  float2* h_fImage = (float2*)malloc(image_memSize);

  //Allocate device output vector for fImage
  float2* d_fImage;
  err = hipMalloc((void **)&d_fImage, image_memSize);
  
  if (err!=hipSuccess){			
    std::cout<<"Failed to allocate device fImage:"<<std::endl; 
    return 1;
  }
  
  // create CUFFT plan
  hipfftHandle plan;
  if (hipfftPlan1d(&plan, nBinsGlobal, HIPFFT_R2C, 1)!=HIPFFT_SUCCESS){
    std::cout<<"FFT plan creation failed"<<std::endl;
    return 1;
  }
  
  //DO THE FOURIER TRANSFORM OF d_sigSum;
  
  //run transform
  if (hipfftExecR2C(plan,d_sigSum_NZ, d_fImage) != HIPFFT_SUCCESS){
    std::cout<<"FFT: ExecR2C failed"<<std::endl;
    return 1;
  }
  //  else 
  
  //copy the output to the host
  
  //  int image_memSize = sizeof(float2) * nBins;
  err= hipMemcpy(h_fImage, d_fImage, image_memSize, hipMemcpyDeviceToHost);	
  
  if (err!=hipSuccess){
    std::cout<<"Failed to copy fImage from device to host"<<std::endl;
    return 1;
  }   
  
  //fill the histogram
  for (int i=0; i<nBinsGlobal; i++){
    float amplitude;
    if (i<floor(nBinsGlobal/2)+1)	{  
      amplitude=sqrt(pow(h_fImage[i].x,2)+pow(h_fImage[i].y,2));
      if (amplitude==amplitude) sumFreq_fImage[iStep]->SetBinContent(i+1,amplitude);
      else std::cout<<"DMScan::sumFrequencies_GPU: nan in FFT bin "<<i<<", iStep: "<<iStep<<std::endl; 
	     //sumFreq_fImage[iStep]->SetBinContent(i+1,amplitude);
    }  
    else{
      int ind=nBinsGlobal-i;
      amplitude=sqrt(pow(h_fImage[ind].x,2)+pow(h_fImage[ind].y,2));
      if (amplitude==amplitude) sumFreq_fImage[iStep]->SetBinContent(i+1,amplitude);
      else std::cout<<"DMScan::sumFrequencies_GPU: nan in FFT bin "<<i<<", iStep: "<<iStep<<std::endl;
    }	
  }

  free(sigSum);
  free(h_sigSum_NZ);
  free(h_fImage);
    
  hipFree(d_fImage);
  hipFree(d_sigSum_NZ);
  hipFree(d_sigSum);
  hipfftDestroy(plan);
  
  return 0;
}

int DMScan::closeGPU()
{
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  // Free device global memory
  err = hipFree(d_sigArray);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to free device input signal vector (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  err = hipDeviceReset();
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return 0;
  //STEP 12
}

int DMScan::initScan(std::string rootfile)
{
  inputFile=new TFile(rootfile.c_str());

  if (inputFile->IsZombie()) {
    std::cout<<"DMScan::initScan root file "<<rootfile.c_str()<<" not found"<<std::endl;
    return 1;
  }
  
  char tmp[100];
  for (int y=0; y<512; y++){
    sprintf(tmp,"sigTimeProfile_freqID_%d",y);
    sigTimeProfile.push_back((TH1F*)inputFile->Get(tmp));
    //sigTimeProfile[y]=(TH1F*)inputFile->Get(tmp);
    means.push_back(0);
    nBins=sigTimeProfile[y]->GetNbinsX();
    
    for (int i=0; i<nBins; i++){
      means[y]+=sigTimeProfile[y]->GetBinContent(i);
    }
    means[y]=means[y]/nBins;
    //      std::cout<<"means:   "<<means[y]<<std::endl;
    if (means[y]!=0) sigTimeProfile[y]->Scale(pow(means[y],-1));
  }
  
  nPeriods=nBins/nBinsPerPeriod;
  
  for (int i=0; i<nPointsToScan; i++){
    sprintf(tmp,"sumFreq_%d",i);
    sumFreq.push_back(new TH1F(tmp,tmp,nBinsGlobal,0,nBinsGlobal));
  }
  
  for (int i=0; i<nPointsToScan; i++){
    sprintf(tmp,"sumFreq_fImage_%d",i);
    sumFreq_fImage.push_back(new TH1F(tmp,tmp,nBinsGlobal,0,nBinsGlobal));
  }
  
  //read file contents into the memory:
  size_t size_input=(nBins*512)*sizeof(float);
  sigArray=(float*)malloc(size_input);
  for (int i = 0; i < 512; ++i){
    for (int j=0; j < nBins; ++j){
	  sigArray[i*nBins+j] = sigTimeProfile[i]->GetBinContent(j+1);
	}
    }
  
  //Device input vector:
  d_sigArray = NULL;
  
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate pinned host memory (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  } 
  
  return 0;
}

