#include "hip/hip_runtime.h"
#include "PFScan.h"
#include "TF1.h"
#include "TStopwatch.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

//before class definition define CUDA kernel func
//see comments on actions in DMScan::sumFrequencies() function
//treat array as 1dim: in thread i sum_j(511-j+i), fullID - ID on a given freq band
__global__ void sumFreq_kernel(float *sigArray, 
			       float *sigSum, 
			       int nFreq, 
			       int nBinsInput, 
			       int nBinsPerPeriod, 
			       float DM, 
			       float l511, 
			       float dL, 
			       float tau, 
			       float period)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  float bico=0;
  int fullID=i;
  for (int iFreq=0; iFreq<nFreq; iFreq++){
    float dT=4.6*(-l511*l511+(l511+iFreq*dL)*(l511+iFreq*dL))*DM*0.001;
    float dTnearest=dT-period*floor(dT/period);
    float delta=dTnearest/tau;
    //QUESTION: WHY %nBins*nFreq???, keep for safety omit few last periods later
    float bico1=sigArray[(((nFreq-1)-iFreq)*nBinsInput
			  +int(floor(fullID+delta)))%(nBinsInput*nFreq)];
    float bico2=sigArray[(((nFreq-1)-iFreq)*nBinsInput
			  +int((floor(fullID+delta)+1)))%(nBinsInput*nFreq)];
    float loFrac=1-((fullID+delta)-floor(fullID+delta));
    float upFrac=1-loFrac;
    bico+=loFrac*bico1+upFrac*bico2;
  }
  sigSum[fullID]=bico;
}

int PFScan::DoScan_GPU(int nThreadsPerBlock)
{
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  // Print the vector length to be used, and compute its size
  size_t size_input=(fNBinsInput*fNFreq)*sizeof(float);
  //  size_t size_output=(fNBins)*sizeof(float);	
  
  // Allocate the device input signal vector 
  err = hipMalloc((void **)&fDev_SigArray, size_input);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate the device input signal vector (error code %s)!\n", hipGetErrorString(err));
    
    //    free(fSigArray);
    
    return 0;
    //exit(EXIT_FAILURE);
  }
  
  // Copy the host input signal vector to the device signal vector in device memory
  err = hipMemcpy(fDev_SigArray, fSigArray, size_input, hipMemcpyHostToDevice);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy input signal vector from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  //  free(fSigArray);

  for (int i=0; i<fNScanPoints; i++){
    if ((i+1)%10==0||i+1==1) std::cout<<"PFScan::DoScan_GPU;  process point "<<i+1<<std::endl;
    DoCompensation_GPU(i, nThreadsPerBlock);
    if (fDoFFT) DoCuFFT(i);
  }

  CloseGPU();

  return 0; 
}


int PFScan::DoCompensation_GPU(int iStep, int nThreadsPerBlock)
{
  TStopwatch stwch;
  float DM=fDM0+fScanStep*iStep;
  
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  size_t size_output=fNBinsInput*sizeof(float);

  //host compensated DM
  float* sigSum=(float *)malloc(size_output);
  
  // device compensated DM
  float* d_sigSum;
  err = hipMalloc((void **)&d_sigSum, size_output);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate compensated signal vector on the device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  // Launch the Vector Add CUDA Kernel
  int nBlocksPerGrid = floor(fNBinsInput/nThreadsPerBlock)+1;
  //  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  //  std::cout<<"parameters: "<<fNFreq<<"  "<<fNBins<<"  "<<DM<<"  "<<fL511<<"  "<<fDL<<"  "<<fTau<<"  "<<fPeriod<<"   blocksPerGrid "<<nBlocksPerGrid<<std::endl;
  sumFreq_kernel<<<nBlocksPerGrid, nThreadsPerBlock>>>
    (fDev_SigArray, d_sigSum, fNFreq, fNBinsInput, 
     fNBinsPerPeriod, DM, fL511, fDL, fTau, fPeriod);
  err = hipGetLastError();
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to launch sumFreq kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  // Copy the device result vector in device memory to the host result vector in host memory.
  //  printf("Copy output data from the CUDA device to the host memory:");
  //  std::cout<<"address: "<<sigSum<<"  "<<sigSum[0]<<"  "<<d_sigSum<<"  "<<size_output<<"  "<<hipMemcpyDeviceToHost<<std::endl;
  err = hipMemcpy(sigSum, d_sigSum, size_output, hipMemcpyDeviceToHost);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy compensated signal vector from device to host (error code: %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  for (int i=0; i<fNBins; i++){
    if (sigSum[i]==sigSum[i]&&i<fNBins) fHCompSig[iStep]->SetBinContent(i+1,sigSum[i]);
    else fHCompSig[iStep]->SetBinContent(i+1,fNFreq);
  }
  
  free(sigSum);
  hipFree(d_sigSum);

  fHCompTiming->Fill(stwch.RealTime(),1);

  return 0;  
}

int PFScan::DoCuFFT(int iStep)
{
  TStopwatch stwch;
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
    //allocate host vector with non-zero elements and image
  //use common number of bins for all runs (fNBins)
  
  // std::cout<<"NBINS: "<<fNBins<<std::endl;

  int signal_memSize=sizeof(float)*fNBins;
  float* h_sigSum_NZ = (float*)malloc(signal_memSize);
  //fill the vector
  for (int i=0; i<fNBins; i++){
    float bico=fHCompSig[iStep]->GetBinContent(i+1);
    if (bico!=0&&bico==bico&&i<fNBins) h_sigSum_NZ[i]=bico;
    else h_sigSum_NZ[i]=fNFreq;
  }
  
  //allocate device vector with non-zero elements:
  int image_memSize = sizeof(float2) * (floor(fNBins/2)+1);
  float* d_sigSum_NZ;
  err=hipMalloc((void **)&d_sigSum_NZ, image_memSize);
  if (err!=hipSuccess){			
    std::cout<<"Failed to allocate device signal NZ vector"<<std::endl; 
    return 1;
  }
  
  //copy host to device
  err = hipMemcpy(d_sigSum_NZ, h_sigSum_NZ, image_memSize, hipMemcpyHostToDevice);

  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy signal NZ vector from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  /////////
  
  //allocate host and device images
  float2* h_fImage = (float2*)malloc(image_memSize);

  //Allocate device output vector for fImage
  float2* d_fImage;
  err = hipMalloc((void **)&d_fImage, image_memSize);
  
  if (err!=hipSuccess){			
    std::cout<<"Failed to allocate device fImage:"<<std::endl; 
    return 1;
  }
  
  // create CUFFT plan
  hipfftHandle plan;
  if (hipfftPlan1d(&plan, fNBins, HIPFFT_R2C, 1)!=HIPFFT_SUCCESS){
    std::cout<<"FFT plan creation failed"<<std::endl;
    return 1;
  }
  
  //DO THE FOURIER TRANSFORM OF d_sigSum;
  
  //run transform
  if (hipfftExecR2C(plan,d_sigSum_NZ, d_fImage) != HIPFFT_SUCCESS){
    std::cout<<"FFT: ExecR2C failed"<<std::endl;
    return 1;
  }
  //  else 
  
  //copy the output to the host
  
  //  int image_memSize = sizeof(float2) * fNBins;
  err= hipMemcpy(h_fImage, d_fImage, image_memSize, hipMemcpyDeviceToHost);	
  
  if (err!=hipSuccess){
    std::cout<<"Failed to copy fImage from device to host"<<std::endl;
    return 1;
  }   
  
  //fill the histogram
  for (int i=0; i<fNBins; i++){
    if (i<floor(fNBins/2)+1)	{  
      float amplitude=sqrt(pow(h_fImage[i].x,2)+pow(h_fImage[i].y,2));
      if (amplitude==amplitude) fHCompSig_FFTImage[iStep]->SetBinContent(i+1,amplitude);
      else std::cout<<"DMScan::sumFrequencies_GPU: nan in FFT bin "
		    <<i<<", iStep: "<<iStep<<std::endl; 
	     //sumFreq_fImage[iStep]->SetBinContent(i+1,amplitude);
    }  
    else{
      int ind=fNBins-i;
      float amplitude=sqrt(pow(h_fImage[ind].x,2)+pow(h_fImage[ind].y,2));
      if (amplitude==amplitude) fHCompSig_FFTImage[iStep]->SetBinContent(i+1,amplitude);
      else std::cout<<"DMScan::sumFrequencies_GPU: nan in FFT bin "
		    <<i<<", iStep: "<<iStep<<std::endl;
    }	
  }

  free(h_sigSum_NZ);
  free(h_fImage);
    
  hipFree(d_fImage);
  hipFree(d_sigSum_NZ);
  hipfftDestroy(plan);

  fHFFTTiming->Fill(stwch.RealTime(),1);

  return 0;
}


int PFScan::CloseGPU()
{
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  // Free device global memory
  err = hipFree(fDev_SigArray);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to free device input signal vector (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  err = hipDeviceReset();
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return 0;
  //STEP 12
}
