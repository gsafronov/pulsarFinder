#include "hip/hip_runtime.h"
#include "DMScan.h"
#include <hip/hip_runtime.h>

//before class definition define CUDA kernel func
//see comments on actions in DMScan::sumFrequencies() function
//treat array as 1dim: in thread i sum_j(511-j+i), fullID - ID on a given freq band
__global__ void sumFreq_kernel(float *sigArray, float *sigSum, int nFreq, int nBins, int nBinsPerPeriod, float DM, float l511, float dL, float tau, float period)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  float bico=0;
  int fullID=i;
  for (int iFreq=0; iFreq<nFreq; iFreq++){
    float dT=4.6*(-l511*l511+(l511+iFreq*dL)*(l511+iFreq*dL))*DM*0.001;
    float dTnearest=dT-period*floor(dT/period);
    float delta=dTnearest/tau;
    float bico1=sigArray[((511-iFreq)*nBins+int(floor(fullID+delta)))%(nBins*nFreq)];
    float bico2=sigArray[((511-iFreq)*nBins+int((floor(fullID+delta)+1)))%(nBins*nFreq)];
    float loFrac=1-((fullID+delta)-floor(fullID+delta));
    float upFrac=1-loFrac;
    bico+=loFrac*bico1+upFrac*bico2;
  }
  sigSum[fullID]=bico;
}

int DMScan::loadDataToGPU()
{
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  // Print the vector length to be used, and compute its size
  size_t size_input=(fNBins*fNFreq)*sizeof(float);
  //  size_t size_output=(fNBins)*sizeof(float);	
  
  // Allocate the device input signal vector 
  err = hipMalloc((void **)&fDev_SigArray, size_input);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate the device input signal vector (error code %s)!\n", hipGetErrorString(err));
    
    free(fSigArray);
    
    return 0;
    //exit(EXIT_FAILURE);
  }
  
  // Copy the host input signal vector to the device signal vector in device memory
  err = hipMemcpy(fDev_SigArray, fSigArray, size_input, hipMemcpyHostToDevice);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy input signal vector from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  free(fSigArray);
  return 0;
}

int DMScan::sumFrequencies_GPU(int iStep)
{
  float DM=fDM0+fScanStep*iStep;
  
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  size_t size_output=fNBins*sizeof(float);

  //host compensated DM
  float* sigSum=(float *)malloc(size_output);
  
  // device compensated DM
  float* d_sigSum;
  err = hipMalloc((void **)&d_sigSum, size_output);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate compensated signal vector on the device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 32;
  int blocksPerGrid = floor(fNBins/threadsPerBlock)+1;
  //  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  std::cout<<"parameters: "<<fNFreq<<"  "<<fNBins<<"  "<<DM<<"  "<<fL511<<"  "<<fDL<<"  "<<fTau<<"  "<<fPeriod<<"   blocksPerGrid "<<blocksPerGrid<<std::endl;
  sumFreq_kernel<<<blocksPerGrid, threadsPerBlock>>>(fDev_SigArray, d_sigSum, fNFreq, fNBins, fNBinsPerPeriod, DM, fL511, fDL, fTau, fPeriod);
  err = hipGetLastError();
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to launch sumFreq kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  // Copy the device result vector in device memory to the host result vector in host memory.
  //  printf("Copy output data from the CUDA device to the host memory:");
  //  std::cout<<"address: "<<sigSum<<"  "<<sigSum[0]<<"  "<<d_sigSum<<"  "<<size_output<<"  "<<hipMemcpyDeviceToHost<<std::endl;
  err = hipMemcpy(sigSum, d_sigSum, size_output, hipMemcpyDeviceToHost);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy compensated signal vector from device to host (error code: %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  for (int i=0; i<fNBins; i++){
    if (sigSum[i]==sigSum[i]&&i<fNBins) sumFreq[iStep]->SetBinContent(i+1,sigSum[i]);
    else sumFreq[iStep]->SetBinContent(i+1,512);
  }
  
  //allocate host vector with non-zero elements and image
  //use common number of bins for all runs (fNBins)
  
  // std::cout<<"NBINS: "<<fNBins<<std::endl;

  int signal_memSize=sizeof(float)*fNBins;
  float* h_sigSum_NZ = (float*)malloc(signal_memSize);
  //fill the vector
  for (int i=0; i<fNBins; i++){
    if (sigSum[i]!=0&&sigSum[i]==sigSum[i]&&i<fNBins) h_sigSum_NZ[i]=sigSum[i];
    else h_sigSum_NZ[i]=512;
  }
  
  //allocate device vector with non-zero elements:
  int image_memSize = sizeof(float2) * (floor(fNBins/2)+1);
  float* d_sigSum_NZ;
  err=hipMalloc((void **)&d_sigSum_NZ, image_memSize);
  if (err!=hipSuccess){			
    std::cout<<"Failed to allocate device signal NZ vector"<<std::endl; 
    return 1;
  }
  
  //copy host to device
  err = hipMemcpy(d_sigSum_NZ, h_sigSum_NZ, image_memSize, hipMemcpyHostToDevice);

  if (err != hipSuccess){
    fprintf(stderr, "Failed to copy signal NZ vector from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  /////////
  
  //allocate host and device images
  float2* h_fImage = (float2*)malloc(image_memSize);

  //Allocate device output vector for fImage
  float2* d_fImage;
  err = hipMalloc((void **)&d_fImage, image_memSize);
  
  if (err!=hipSuccess){			
    std::cout<<"Failed to allocate device fImage:"<<std::endl; 
    return 1;
  }
  
  // create CUFFT plan
  hipfftHandle plan;
  if (hipfftPlan1d(&plan, fNBins, HIPFFT_R2C, 1)!=HIPFFT_SUCCESS){
    std::cout<<"FFT plan creation failed"<<std::endl;
    return 1;
  }
  
  //DO THE FOURIER TRANSFORM OF d_sigSum;
  
  //run transform
  if (hipfftExecR2C(plan,d_sigSum_NZ, d_fImage) != HIPFFT_SUCCESS){
    std::cout<<"FFT: ExecR2C failed"<<std::endl;
    return 1;
  }
  //  else 
  
  //copy the output to the host
  
  //  int image_memSize = sizeof(float2) * fNBins;
  err= hipMemcpy(h_fImage, d_fImage, image_memSize, hipMemcpyDeviceToHost);	
  
  if (err!=hipSuccess){
    std::cout<<"Failed to copy fImage from device to host"<<std::endl;
    return 1;
  }   
  
  //fill the histogram
  for (int i=0; i<fNBins; i++){
    float amplitude;
    if (i<floor(fNBins/2)+1)	{  
      amplitude=sqrt(pow(h_fImage[i].x,2)+pow(h_fImage[i].y,2));
      if (amplitude==amplitude) sumFreq_fImage[iStep]->SetBinContent(i+1,amplitude);
      else std::cout<<"DMScan::sumFrequencies_GPU: nan in FFT bin "<<i<<", iStep: "<<iStep<<std::endl; 
	     //sumFreq_fImage[iStep]->SetBinContent(i+1,amplitude);
    }  
    else{
      int ind=fNBins-i;
      amplitude=sqrt(pow(h_fImage[ind].x,2)+pow(h_fImage[ind].y,2));
      if (amplitude==amplitude) sumFreq_fImage[iStep]->SetBinContent(i+1,amplitude);
      else std::cout<<"DMScan::sumFrequencies_GPU: nan in FFT bin "<<i<<", iStep: "<<iStep<<std::endl;
    }	
  }

  free(sigSum);
  free(h_sigSum_NZ);
  free(h_fImage);
    
  hipFree(d_fImage);
  hipFree(d_sigSum_NZ);
  hipFree(d_sigSum);
  hipfftDestroy(plan);
  
  return 0;
}

int DMScan::closeGPU()
{
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  
  // Free device global memory
  err = hipFree(fDev_SigArray);
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to free device input signal vector (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  err = hipDeviceReset();
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return 0;
  //STEP 12
}

